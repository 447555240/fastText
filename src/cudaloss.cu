#include "hip/hip_runtime.h"
#include <iostream>
#include <mutex>
#include "cudaloss.h"
#include "matrix.h"
#include "densematrix.h"
#include "vector.h"

namespace fasttext {
static const float one = 1.0;
static const float zero = 0.0;
static const real epsilon = 0.00001f;
real* CudaSoftmaxLoss::d_wo_;

#define CUDA_CHECK(error) { \
  if (error!=hipSuccess){ \
    std::cerr<<"CUDA ERROR "<< hipGetErrorName(error) << " in file "  << __FILE__ << " line " <<__LINE__<< std::endl; \
    exit(0); \
  }  \
}

static const char* cublasGetErrorEnum(hipblasStatus_t error)
{
  switch (error)
  {
    case HIPBLAS_STATUS_SUCCESS:
      return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:
      return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:
      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:
      return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:
      return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:
      return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:
      return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:
      return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:
      return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:
      return "HIPBLAS_STATUS_UNKNOWN";
    default:
      return "UNKNOWN CUBLAS ERROR";
  }
}

#define CUBLAS_CHECK(err) \
{ \
  if (HIPBLAS_STATUS_SUCCESS != err) { \
    std::cerr<<"CUBLAS ERROR "<< cublasGetErrorEnum(err) << " in file "  << __FILE__ << " line " <<__LINE__<< std::endl; \
    exit(0); \
  } \
}

CudaState::CudaState(int32_t hiddenSize, int32_t outputSize, int32_t seed, uint32_t batchSize):Model::State(hiddenSize, outputSize, seed),
	        targets(batchSize), lrs(batchSize), hiddens(batchSize, Vector(hiddenSize)), inputs(batchSize), batchIndex(0), maxBatchSize(batchSize) {
  int64_t M = outputSize;
  int64_t N = hiddenSize;
  CUDA_CHECK(hipMalloc((void**)&d_hidden_, batchSize*N*sizeof(real)));
  CUDA_CHECK(hipMalloc((void**)&d_output_, M*sizeof(real)));
  CUDA_CHECK(hipMalloc((void**)&d_softmax_output_, M*sizeof(real)));
  CUDA_CHECK(hipMalloc((void**)&d_output_diff_, M*sizeof(real)));
  CUDA_CHECK(hipMalloc((void**)&d_grads_, batchSize*N*sizeof(real)+batchSize*sizeof(real)));  // size of grads + loss
  d_lossValues_ = d_grads_ + batchSize*N;

  stream_ = hipStreamPerThread;
  hipdnnCreate(&cudnn_);
  hipdnnCreateTensorDescriptor(&cudnn_output_desc_);
  hipdnnSetTensor4dDescriptor(cudnn_output_desc_, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, 1, 1, M);
  hipdnnSetStream(cudnn_, stream_);
  CUBLAS_CHECK(hipblasCreate(&cublas_));
  CUBLAS_CHECK(hipblasSetStream(cublas_, stream_));
}

CudaState::~CudaState() {
  CUDA_CHECK(hipFree(d_hidden_));
  CUDA_CHECK(hipFree(d_output_));
  CUDA_CHECK(hipFree(d_softmax_output_));
  CUDA_CHECK(hipFree(d_output_diff_));
  CUDA_CHECK(hipFree(d_grads_));
  hipdnnDestroyTensorDescriptor(cudnn_output_desc_);
  hipdnnDestroy(cudnn_);
  CUBLAS_CHECK(hipblasDestroy(cublas_));
}

void CudaState::addToBatch(int32_t target, real lr, const std::vector<int32_t>& input) {
  targets[batchIndex] = target;
  lrs[batchIndex] = lr;
  hiddens[batchIndex] = hidden;
  inputs[batchIndex] = input;
  batchIndex++;
}

CudaSoftmaxLoss::CudaSoftmaxLoss(std::shared_ptr<Matrix>& wi, std::shared_ptr<Matrix>& wo, bool normalizeGradient):SoftmaxLoss(wo), wi_(wi), normalizeGradient_(normalizeGradient) {
}

CudaSoftmaxLoss::~CudaSoftmaxLoss() {
}

bool CudaSoftmaxLoss::init() {
  // Copy wo from host to device
  int64_t m = wo_->size(0);
  int64_t n = wo_->size(1);
  std::vector<real> tmpwo(m*n);
  real* pBegin = tmpwo.data();
  for( int64_t i=0; i<m; i++ ) {
    Vector v(n);
    wo_->addRowToVector(v, i);
    memcpy(pBegin+i*n, v.data(), n*sizeof(real));
  }

  CUDA_CHECK(hipMalloc((void**)&d_wo_, m*n*sizeof(real)));
  CUDA_CHECK(hipMemcpy(d_wo_, pBegin, m*n*sizeof(real), hipMemcpyHostToDevice));
  return true;
}

void CudaSoftmaxLoss::shutdown() {
  // Copy wo from device back to host
  int64_t m = wo_->size(0);
  int64_t n = wo_->size(1);
  std::vector<real> tmpwo(m*n);
  CUDA_CHECK(hipMemcpy(tmpwo.data(), d_wo_, m*n*sizeof(real), hipMemcpyDeviceToHost));
  real* pBegin = tmpwo.data();
  for( int64_t i=0; i<m; i++ ) {
    Vector clear(n);
    wo_->addRowToVector(clear, i);
    wo_->addVectorToRow(clear, i, -1.0);

    Vector add(n);
    memcpy(add.data(), pBegin+i*n, n*sizeof(real));
    wo_->addVectorToRow(add, i, 1.0);
  }
  CUDA_CHECK(hipFree(d_wo_));
}

real CudaSoftmaxLoss::forward(
      const std::vector<int32_t>& targets,
      int32_t targetIndex,
      Model::State& state,
      real lr,
      bool backprop) {
  throw std::runtime_error("forward is not supported in CudaSoftmaxLoss");
}

void CudaSoftmaxLoss::computeOutput(Model::State& state) const {
}

bool CudaSoftmaxLoss::batchforward_enabled() const {
  return true;
}

__global__
void CudacomputeDiff(real* softmax_output, size_t output_n, real* output_diff, real* loss, int32_t target, real lr) {
  int output_idx = blockIdx.x*blockDim.x + threadIdx.x;

  if( threadIdx.x==0 && blockIdx.x==0 ) {
    *loss = softmax_output[target];
  }

  if( output_idx < output_n ) {
    real label = (output_idx==target)?1.0:0.0;
    output_diff[output_idx] = lr * (label - softmax_output[output_idx]);
  }
}

void CudaSoftmaxLoss::forward2batch(int32_t target, Model::State& state, real lr, bool backprop, bool normalizeGradient, const std::vector<int32_t>& input) {
  CudaState& batchState = static_cast<CudaState&>(state);

#ifdef FASTTEXT_CUDA_DEBUG
  Model::State CpuState(state);
  compare(CpuState, batchState, true, false);
  SoftmaxLoss::computeOutput(CpuState);
  if (backprop) {
    int32_t osz = wo_->size(0);
    for (int32_t i = 0; i < osz; i++) {
      real label = (i == target) ? 1.0 : 0.0;
      real alpha = lr * (label - CpuState.output[i]);
      CpuState.grad.addRow(*wo_, i, alpha);
      wo_->addVectorToRow(CpuState.hidden, i, alpha);
    }
  }
  real cpuloss = -SoftmaxLoss::log(CpuState.output[target]);
#endif

  batchState.addToBatch(target, lr, input);
  if( batchState.batchIndex==batchState.maxBatchSize ) {
    flush(batchState, backprop);
  }

#ifdef FASTTEXT_CUDA_DEBUG
  if( backprop )
    compare(CpuState, batchState, false, true);
#endif  
}

void CudaSoftmaxLoss::flush(Model::State& state, bool backprop) {
  CudaState& batchState = static_cast<CudaState&>(state);
  std::vector<real> lossValues(batchState.batchIndex);
  std::vector<Vector> grads(batchState.batchIndex, Vector(batchState.grad.size()));
  batchforward(batchState, batchState.batchIndex, backprop, lossValues, grads);
  for (const auto& lossValue : lossValues) {
    batchState.incrementNExamples(lossValue);
  }
  int idx = 0;
  for (auto& grad : grads) {
    if (normalizeGradient_) {
      grad.mul(1.0 / batchState.inputs[idx].size());
    }
    for (auto it = batchState.inputs[idx].cbegin(); it != batchState.inputs[idx].cend(); ++it) {
      wi_->addVectorToRow(grad, *it, 1.0);
    }
    idx++;
  }
  batchState.batchIndex = 0;
}

void CudaSoftmaxLoss::compare(const Model::State& CPUState, const CudaState& GPUState, bool CmpWo, bool CmpSoftmaxOutput) {
  if( CmpWo ) {
    int64_t m = wo_->size(0);
    int64_t n = wo_->size(1);
    DenseMatrix* wo = dynamic_cast<DenseMatrix*>(wo_.get());
    std::vector<real> tmpwo(m*n);
    CUDA_CHECK(hipMemcpy(tmpwo.data(), d_wo_, m*n*sizeof(real), hipMemcpyDeviceToHost));
    for( int64_t i=0; i<m; i++ ) {
      for( int64_t j=0; j<n; j++ ) {
        if( fabs(tmpwo[i*n+j]-wo->at(i,j))>epsilon )
          printf("\nwo[%ld,%ld] not match %f %f\n", i, j, tmpwo[i*n+j], wo->at(i,j));
      }
    }
  }
  if( CmpSoftmaxOutput ) {
    int64_t m = wo_->size(0);
    std::vector<real> tmpSoftMax(m);
    CUDA_CHECK(hipMemcpy(tmpSoftMax.data(), GPUState.d_softmax_output_, m*sizeof(real), hipMemcpyDeviceToHost));
    for( int64_t i=0; i<m; i++ ) {
      if( fabs(tmpSoftMax[i]-CPUState.output[i])>epsilon )
	printf("\nsoftmax [%ld] not match %f %f\n", i, tmpSoftMax[i], CPUState.output[i]);
    }
  }
}

void CudaSoftmaxLoss::batchforward(
      CudaState& batchState,
      uint32_t batchSize,
      bool backprop,
      std::vector<real>& lossValues,
      std::vector<Vector>& grads) {
  int M = wo_->size(0);  // labels
  int N = wo_->size(1);  // dims

  // Copy hiddens -> d_hidden_
  std::vector<real> h_hiddens(batchSize*N);
  real* pHidden = h_hiddens.data();
  for(uint32_t i=0; i<batchSize; i++ ) {
    memcpy(pHidden+i*N, batchState.hiddens[i].data(), N*sizeof(real));
  }
  CUDA_CHECK(hipMemcpy(batchState.d_hidden_, pHidden, batchSize*N*sizeof(real), hipMemcpyHostToDevice));
  h_hiddens.clear();

  for(uint32_t i=0; i<batchSize; i++ ) {
    // compute output
    CUBLAS_CHECK(hipblasSgemv(batchState.cublas_, HIPBLAS_OP_T,
      N, M,
      &one,
      d_wo_, N,
      batchState.d_hidden_+N*i, 1,
      &zero,
      batchState.d_output_, 1));

    // compute softmax
    hipdnnSoftmaxForward(batchState.cudnn_, hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_ACCURATE, hipdnnSoftmaxMode_t::HIPDNN_SOFTMAX_MODE_INSTANCE,
      &one, batchState.cudnn_output_desc_, batchState.d_output_,
      &zero, batchState.cudnn_output_desc_, batchState.d_softmax_output_);  

    // compute loss
    CudacomputeDiff<<<(M+255)/256, 256, 0, batchState.stream_>>>(
      batchState.d_softmax_output_,
      M,
      batchState.d_output_diff_,
      batchState.d_lossValues_ + i,
      batchState.targets[i], batchState.lrs[i]);

    if( backprop ) {
      // compute grad
      CUBLAS_CHECK(hipblasSgemv(batchState.cublas_, HIPBLAS_OP_T,
        M, N,
        &one,
        d_wo_, M,
        batchState.d_output_diff_, 1,
        &zero,
        batchState.d_grads_+N*i, 1));

      // update wo
      CUBLAS_CHECK(hipblasSger(batchState.cublas_,
        N, M,
        &one,
        batchState.d_hidden_, 1,
        batchState.d_output_diff_, 1,
        d_wo_, N));
    }
  }

  hipStreamSynchronize(batchState.stream_);

  // Copy d_lossValues_ -> lossValues, d_grads_ -> grads
  std::vector<real> h_grads_loss(batchSize*N + batchSize);
  CUDA_CHECK(hipMemcpy(h_grads_loss.data(), batchState.d_grads_, batchSize*N*sizeof(real)+batchSize*sizeof(real), hipMemcpyDeviceToHost));
  real* pGrad = h_grads_loss.data();
  for(uint32_t i=0; i<batchSize; i++ ) {
    memcpy(grads[i].data(), pGrad+i*N, N*sizeof(real));
    lossValues[i] = -Loss::log(h_grads_loss[batchSize*N+i]);
  }
}

} // namespace fasttext
